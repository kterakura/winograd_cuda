﻿#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <random>

#define SIZE  (32*32*16)
#define PSIZE  (34*34*16)

void initialData( signed char *a, int size){
    for (int i = 0; i < size; i++) a[i] =  i;
    return;
}

__device__  signed char clamp(int v)
{
	if(v <= -128) return -128;
	else if(v > 127) return 127;
    else return v;
}


__global__ void conv( signed char *input,  signed char *filter,  signed char *output){
    // input = (32,16,16), output =(32,16,16), filter =(3,3,16,16), s_input = (34,18,16)

    const int id = threadIdx.x + blockDim.x*threadIdx.y, ch_id = blockIdx.x, block_size = blockDim.x*blockDim.y;
    const int conv_place = (ch_id&1), conv_ch = (ch_id>>1);
    
    __shared__  signed char s_input[9792]; //34*18*16
	__shared__  signed char s_filter[144]; //3*3*16
    __shared__ int s_output[512];  //32*16
    
    // init shared memory
    for(int i = id; i < 9792; i+=block_size) s_input[i] = 0;
    for(int j = id; j < 144; j+=block_size) s_filter[j] = filter[j + conv_ch*144];
    for(int k = id; k < 512; k+=block_size) s_output[k] = 0;

    for (int i = id; i < 8704; i+=blockDim.x){   //8704 = 32*(16+1)*16,   544 = 32*(16+1)
        const int x = i&31, y = ((i%544)>>5), ch = i/(544);
        s_input[(x+1) + 34*(y+(conv_place^1)) + 612*ch] = input[x + ((y + (conv_place<<4) - conv_place)<<5) + (ch<<10)];
    }__syncthreads();

    for (int n = id; n < 8192; n += block_size){  //9782 = output_2d * input_ch
        const int x = n&31;
        const int y = (n&511)>>5;
        const int ch = n>>9;
        const int x0 = s_input[(x)  +34*(y)  + 612*ch] * s_filter[0 + 9*ch];
        const int x1 = s_input[(x+1)+34*(y)  + 612*ch] * s_filter[1 + 9*ch];
        const int x2 = s_input[(x+2)+34*(y)  + 612*ch] * s_filter[2 + 9*ch];
        const int x3 = s_input[(x)  +34*(y+1)+ 612*ch] * s_filter[3 + 9*ch];
        const int x4 = s_input[(x+1)+34*(y+1)+ 612*ch] * s_filter[4 + 9*ch];
        const int x5 = s_input[(x+2)+34*(y+1)+ 612*ch] * s_filter[5 + 9*ch];
        const int x6 = s_input[(x)  +34*(y+2)+ 612*ch] * s_filter[6 + 9*ch];
        const int x7 = s_input[(x+1)+34*(y+2)+ 612*ch] * s_filter[7 + 9*ch];
        const int x8 = s_input[(x+2)+34*(y+2)+ 612*ch] * s_filter[8 + 9*ch];
        atomicAdd(&s_output[x+(y<<5)], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }

    __syncthreads();
    for (int i = id; i < 512; i+=blockDim.x){
        const int x = i&31, y = (i>>5) + (conv_place<<4);
        output[x + (y<<5) + (conv_ch<<10)] = clamp(((s_output[i] + (1 << 4)) >>5)) + 128;
    }
}


__global__ void winograd( signed char *input,  signed short *weight,  signed char *output){
	// dim3(16) dim3(4,4,16)
    const int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z, bx = blockIdx.x;
    // const int id = threadIdx.x + (threadIdx.y<<2) + (threadIdx.z<<4);
    const int tx_ty = threadIdx.x + (threadIdx.y<<2);
	__shared__ signed char input_smem[16][16][16];
	__shared__ int BtdB[16][16][16];
	__shared__ int I[16][16][4][4];
	
    // input to smem
    for(int i=0; i<16; i++){
        const int in_start = (i<<1)+tx + ((bx<<1)+ty)*34 + tz*1156;
        input_smem[i][tz][tx_ty] = input[in_start];
        I[i][tz][ty][tx] = 0;
    }
    __syncthreads();

    BtdB[tz][tx_ty][0] = input_smem[tz][tx_ty][0]-input_smem[tz][tx_ty][8]-input_smem[tz][tx_ty][2]+input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][1] = input_smem[tz][tx_ty][1]-input_smem[tz][tx_ty][9]+input_smem[tz][tx_ty][2]-input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][2] = -input_smem[tz][tx_ty][1]+input_smem[tz][tx_ty][9]+input_smem[tz][tx_ty][2]-input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][3] = input_smem[tz][tx_ty][1]-input_smem[tz][tx_ty][9]-input_smem[tz][tx_ty][3]+input_smem[tz][tx_ty][11];
    BtdB[tz][tx_ty][4] = input_smem[tz][tx_ty][4]+input_smem[tz][tx_ty][8]-input_smem[tz][tx_ty][6]-input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][5] = input_smem[tz][tx_ty][5]+input_smem[tz][tx_ty][9]+input_smem[tz][tx_ty][6]+input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][6] = -input_smem[tz][tx_ty][5]-input_smem[tz][tx_ty][9]+input_smem[tz][tx_ty][6]+input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][7] = input_smem[tz][tx_ty][5]+input_smem[tz][tx_ty][9]-input_smem[tz][tx_ty][7]-input_smem[tz][tx_ty][11];
    BtdB[tz][tx_ty][8] = -input_smem[tz][tx_ty][4]+input_smem[tz][tx_ty][8]+input_smem[tz][tx_ty][6]-input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][9] = -input_smem[tz][tx_ty][5]+input_smem[tz][tx_ty][9]-input_smem[tz][tx_ty][6]+input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][10] = input_smem[tz][tx_ty][5]-input_smem[tz][tx_ty][9]-input_smem[tz][tx_ty][6]+input_smem[tz][tx_ty][10];
    BtdB[tz][tx_ty][11] = -input_smem[tz][tx_ty][5]+input_smem[tz][tx_ty][9]+input_smem[tz][tx_ty][7]-input_smem[tz][tx_ty][11];
    BtdB[tz][tx_ty][12] = input_smem[tz][tx_ty][4]-input_smem[tz][tx_ty][12]-input_smem[tz][tx_ty][6]+input_smem[tz][tx_ty][14];
    BtdB[tz][tx_ty][13] = input_smem[tz][tx_ty][5]-input_smem[tz][tx_ty][13]+input_smem[tz][tx_ty][6]-input_smem[tz][tx_ty][14];
    BtdB[tz][tx_ty][14] = -input_smem[tz][tx_ty][5]+input_smem[tz][tx_ty][13]+input_smem[tz][tx_ty][6]-input_smem[tz][tx_ty][14];
    BtdB[tz][tx_ty][15] = input_smem[tz][tx_ty][5]-input_smem[tz][tx_ty][13]-input_smem[tz][tx_ty][7]+input_smem[tz][tx_ty][15];
    __syncthreads();

    for(int i=0; i<16; i++){
        for(int k=0; k<16; k++){
            atomicAdd(&I[i][k][ty][tx], BtdB[k][tz][tx_ty]*weight[tx_ty+(tz<<4)+(i<<8)]);
        }
    }
    // for(int i=id; i<65536; i+=256){
        // int ch = i>>12, b_x = (i&4095)>>8;
        // atomicAdd(&I[ch][b_x][ty][tx], BtdB[b_x][tz][tx_ty]*weight[tx_ty+(tz<<4)+(ch<<8)]);
    // }
    __syncthreads();
    // if(bx==0&&tz==0) printf("%d ",I[0][0][ty][tx]);
    const int out_start1 = ((tx_ty<<1)+1) + (((bx<<1)+1)*34) + ((tz)*1156);
    const int out_start2 = ((tx_ty<<1)+2) + (((bx<<1)+1)*34) + ((tz)*1156);
    const int out_start3 = ((tx_ty<<1)+1) + (((bx<<1)+2)*34) + ((tz)*1156);
    const int out_start4 = ((tx_ty<<1)+2) + (((bx<<1)+2)*34) + ((tz)*1156);
    output[out_start1] = clamp((((I[tz][tx_ty][0][0] + I[tz][tx_ty][0][1] + I[tz][tx_ty][0][2] + I[tz][tx_ty][1][0] + I[tz][tx_ty][1][1] + I[tz][tx_ty][1][2] + I[tz][tx_ty][2][0] + I[tz][tx_ty][2][1] + I[tz][tx_ty][2][2]) + (1 << 6)) >>7)) + 128;
    output[out_start2] = clamp((((I[tz][tx_ty][0][1] - I[tz][tx_ty][0][2] - I[tz][tx_ty][0][3] + I[tz][tx_ty][1][1] - I[tz][tx_ty][1][2] - I[tz][tx_ty][1][3] + I[tz][tx_ty][2][1] - I[tz][tx_ty][2][2] - I[tz][tx_ty][2][3]) + (1 << 6)) >>7)) + 128;
    output[out_start3] = clamp((((I[tz][tx_ty][1][0] + I[tz][tx_ty][1][1] + I[tz][tx_ty][1][2] - I[tz][tx_ty][2][0] - I[tz][tx_ty][2][1] - I[tz][tx_ty][2][2] - I[tz][tx_ty][3][0] - I[tz][tx_ty][3][1] - I[tz][tx_ty][3][2]) + (1 << 6)) >>7)) + 128;
    output[out_start4] = clamp((((I[tz][tx_ty][1][1] - I[tz][tx_ty][1][2] - I[tz][tx_ty][1][3] - I[tz][tx_ty][2][1] + I[tz][tx_ty][2][2] + I[tz][tx_ty][2][3] - I[tz][tx_ty][3][1] + I[tz][tx_ty][3][2] + I[tz][tx_ty][3][3]) + (1 << 6)) >>7)) + 128;

}

__global__ void padding( signed char *input,  signed char *output){
    const int id  = threadIdx.x + blockDim.x*threadIdx.y;
    const int idx = threadIdx.x;
    const int idy = threadIdx.y;
    const int ch = blockIdx.x;
    __shared__  signed char s_output[34*34];

    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) s_output[i] = 0;
    __syncthreads();
    s_output[(idx+1) + (idy+1)*34] = input[idx + idy*32 + ch*32*32];
    __syncthreads();
    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) output[i + ch*34*34] = s_output[i];
}


int main(){
    hipEvent_t start, stop;
    float elapsed_time_ms1, elapsed_time_ms2;
    signed char *h_char = ( signed char *)malloc(SIZE * sizeof( signed char));
    initialData(h_char, SIZE);

    // allocate global memory
    signed char *d_char, *d_char_out, *d_char_outp, *d_charp, *d_filter;
    signed short *d_wino;
    hipMalloc( (void **) &d_char, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_outp, 34*34*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_charp, 34*34*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_filter, 3*3*16*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_wino, 4*4*16*16 * sizeof( signed short ) );

    hipMemcpy( d_char, h_char, SIZE * sizeof( signed char), hipMemcpyHostToDevice );

    signed char f;
    signed short f_short;
	FILE* fp;
    signed char x1_1[16*16*3*3];
    signed short wino[16*16*4*4];
    fp = fopen( "./params/layer1.0.conv1.weight", "rb" );
    if (!fp) printf("x1_1: pathを間違えています\n");
    for(int i=0; i<16*16*3*3; i++){
        if( fread( &f, sizeof(f), 1, fp ) < 1 ){
            fputs( "x1_1: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        x1_1[i] = f;
    }
    hipMemcpy(d_filter, x1_1, sizeof( signed char) * 16*16*3*3, hipMemcpyHostToDevice);
    if (fp) fclose(fp);


    fp = fopen( "./wino_params_short/layer1.0.conv1.weight", "rb" );
    if (!fp) printf("wino: pathを間違えています\n");
    for(int i=0; i<16*16*4*4; i++){
        if( fread( &f_short, sizeof(f_short), 1, fp ) < 1 ){
            fputs( "wino: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        wino[i] = f_short;
    }
    hipMemcpy(d_wino, wino, sizeof(signed short) * 16*16*4*4, hipMemcpyHostToDevice);
    if (fp) fclose(fp);
    
    //Measure
    padding<<<16, dim3(32,32)>>>(d_char, d_charp);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMemset(&d_char_outp, 0, sizeof(signed char)*(34*34*16));
    winograd<<<dim3(16), dim3(4,4,16)>>>(d_charp, d_wino, d_char_outp);
    elapsed_time_ms2=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms2, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("winograd:%f\n", elapsed_time_ms2);

    //Measure
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    conv<<<16*2, 256>>>(d_char, d_filter, d_char_out);
    elapsed_time_ms1=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("normal:%f\n", elapsed_time_ms1);



    //check result
    signed char res[32*32*16];
    hipMemcpy(res, d_char_out, sizeof( signed char) * 32*32*16, hipMemcpyDeviceToHost);
    signed char res1[34*34*16];
    hipMemcpy(res1, d_char_outp, sizeof(signed char) * 34*34*16, hipMemcpyDeviceToHost);

    signed char resp[PSIZE] = {0};
    for(int i=0;i<16;i++){
        for (int j=0;j<32; j++){
            for (int k=0;k<32; k++){
                resp[j+1 + (k+1)*34 + i*1156] = res[j + k*32 + i*1024];
            }
        }
    }
    for(int i=34;i<50; i++) printf("%d ", resp[i]);
    printf("\n");
    for(int i=34;i<50; i++) printf("%d ", res1[i]);
    printf("\n");

    int miss = 0;
    // for(int i=0;i<PSIZE; i++) if(resp[i] != res1[i]) {printf("%d ", i); miss++;}
    for(int i=0;i<PSIZE; i++) if(resp[i] != res1[i]) {miss++;}
    if(miss == 0) printf("%f 倍速くなりました。", elapsed_time_ms1/elapsed_time_ms2);
    else if(miss != 0) printf("bat! miss=%d", miss);

    return 0;
    
}