#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include <iostream>
#include <random>

#define SIZE  (32*32*16)

void initialData( signed char *a, int size){
    for (int i = 0; i < size; i++) a[i] =  signed char(i);
    return;
}

__device__  signed char clamp(int v)
{
	if(v <= -128) return -128;
	else if(v > 127) return 127;
    else return v;
}





__global__ void conv( signed char *input,  signed char *filter,  signed char *output){
    // input = (32,16,16), output =(32,16,16), filter =(3,3,16,16), s_input = (34,18,16)

    const int id = threadIdx.x + blockDim.x*threadIdx.y, ch_id = blockIdx.x, block_size = blockDim.x*blockDim.y;
    const int conv_place = (ch_id&1), conv_ch = (ch_id>>1);
    
    __shared__  signed char s_input[9792]; //34*18*16
	__shared__  signed char s_filter[144]; //3*3*16
    __shared__ int s_output[512];  //32*16
    
    // init shared memory
    for(int i = id; i < 9792; i+=block_size) s_input[i] = 0;
    for(int j = id; j < 144; j+=block_size) s_filter[j] = filter[j + conv_ch*144];
    for(int k = id; k < 512; k+=block_size) s_output[k] = 0;

    for (int i = id; i < 8704; i+=blockDim.x){   //8704 = 32*(16+1)*16,   544 = 32*(16+1)
        const int x = i&31, y = ((i%544)>>5), ch = i/(544);
        s_input[(x+1) + 34*(y+(conv_place^1)) + 612*ch] = input[x + ((y + (conv_place<<4) - conv_place)<<5) + (ch<<10)];
    }__syncthreads();

    for (int n = id; n < 8192; n += block_size){  //9782 = output_2d * input_ch
        const int x = n&31;
        const int y = (n&511)>>5;
        const int ch = n>>9;
        const int x0 = s_input[(x)  +34*(y)  + 612*ch] * s_filter[0 + 9*ch];
        const int x1 = s_input[(x+1)+34*(y)  + 612*ch] * s_filter[1 + 9*ch];
        const int x2 = s_input[(x+2)+34*(y)  + 612*ch] * s_filter[2 + 9*ch];
        const int x3 = s_input[(x)  +34*(y+1)+ 612*ch] * s_filter[3 + 9*ch];
        const int x4 = s_input[(x+1)+34*(y+1)+ 612*ch] * s_filter[4 + 9*ch];
        const int x5 = s_input[(x+2)+34*(y+1)+ 612*ch] * s_filter[5 + 9*ch];
        const int x6 = s_input[(x)  +34*(y+2)+ 612*ch] * s_filter[6 + 9*ch];
        const int x7 = s_input[(x+1)+34*(y+2)+ 612*ch] * s_filter[7 + 9*ch];
        const int x8 = s_input[(x+2)+34*(y+2)+ 612*ch] * s_filter[8 + 9*ch];
        atomicAdd(&s_output[x+(y<<5)], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }

    __syncthreads();
    for (int i = id; i < 512; i+=blockDim.x){
        const int x = i&31, y = (i>>5) + (conv_place<<4);
        output[x + (y<<5) + (conv_ch<<10)] = clamp(((s_output[i] + (1 << 4)) >>5)) + 128;
    }
}


__global__ void winograd( signed char *input,  signed short *weight,  signed char *output){
	// dim3(32/2, 32/2) dim3(4,4,16)
    const int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z, bx = blockIdx.x, by = blockIdx.y;
	const int in_start = bx*2 + tx + (by*2+ty)*34 + tz*1156;  //1156 = 34*34
	const int out_start = (bx<<1)+tx + (((by<<1)+ty)<<5) + (tz<<10);  //1024 = 32*32

	// dim3(32/2, 32/2, 16) dim3(16,4,4)
	// const int in_start = tx + ((ty + (bx<<1))<<4) + (tz + (by<<1))*544;  //1156 = 34*34


	__shared__ signed char input_smem [16][4][4];
	__shared__ int Btd [16][4][4];
	__shared__ int BtdB [16][4][4];
	__shared__ int AtI [16][2][4];
	__shared__ int I [16][4][4];
	
	I[tz][ty][tx] = 0;
	input_smem[tz][ty][tx] = input[in_start];
	// __syncthreads();
	switch (ty)
	{
	case 0:
		Btd [tz][ty][tx] = input_smem[tz][tx][0] - input_smem[tz][tx][2];
		break;
	case 1:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 2:
		Btd [tz][ty][tx] = - input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 3:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] - input_smem[tz][tx][3];
		break;
	}
	// __syncthreads();
	switch (tx)
	{
	case 0:
		BtdB[tz][tx][ty] = Btd[tz][ty][0] - Btd[tz][ty][2];
		break;
	case 1:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 2:
		BtdB[tz][tx][ty] = - Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 3:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] - Btd[tz][ty][3];
		break;
	}
	// __syncthreads();
    const int id = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
	for(int i=id; i<4*4*16*16; i+=blockDim.x*blockDim.y*blockDim.z){
        const int ch = i>>8;
        // const int z = ch>>4;
        // const int x = i&3;
        // const int y = (i&15)>>2;
		atomicAdd(&I[ch][ty][tx], BtdB[tz][ty][tx]*weight[i]);
	}
    __syncthreads();

	// for(int i=0; i<16; i++){
	// 	const int w_start = tx + (ty<<2)+ (tz<<4) + (i*16*16);  //256 = 4*4*16
	// 	atomicAdd(&I[i][ty][tx], BtdB[tz][ty][tx]*weight[w_start]);
	// }
	// __syncthreads();

	if(ty > 1) return;
	switch (ty)
	{
	case 0:
		AtI[tz][ty][tx] = I[tz][0][tx] + I[tz][1][tx] + I[tz][2][tx];
		break;
	case 1:
		AtI[tz][ty][tx] = I[tz][1][tx] - I[tz][2][tx] - I[tz][3][tx];
		break;
	}
	// __syncthreads();

	if(tx > 1) return;
	switch (tx)
	{
	case 0:
		output[out_start] = clamp((((AtI[tz][ty][0] + AtI[tz][ty][1] + AtI[tz][ty][2]) + (1 << 6)) >>7)) + 128;
		break;
	case 1:
		output[out_start] = clamp((((AtI[tz][ty][1] - AtI[tz][ty][2] - AtI[tz][ty][3]) + (1 << 6)) >>7)) + 128;
		break;
	}
	// __syncthreads();
	// output[out_start] = clamp(((output_smem[tz][ty][tx] + (1 << 4)) >>5)) + 128;
}

__global__ void padding( signed char *input,  signed char *output){
    const int id  = threadIdx.x + blockDim.x*threadIdx.y;
    const int idx = threadIdx.x;
    const int idy = threadIdx.y;
    const int ch = blockIdx.x;
    __shared__  signed char s_output[34*34];

    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) s_output[i] = 0;
    __syncthreads();
    s_output[(idx+1) + (idy+1)*34] = input[idx + idy*32 + ch*32*32];
    __syncthreads();
    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) output[i + ch*34*34] = s_output[i];
}


int main(){
    hipEvent_t start, stop;
    float elapsed_time_ms;
    signed char *h_char = ( signed char *)malloc(SIZE * sizeof( signed char));
    signed char *h_filter = ( signed char *)malloc(9*16*16 * sizeof( signed char));
    signed short *h_wino = ( signed short *)malloc(16*16*16 * sizeof( signed short));

    initialData(h_char, SIZE);

	// for(int i=0;i<SIZE;i++){
	// 	if(i%32 == 0) printf("\n");
	// 	if(i%1024 == 0) printf("\n");
	// 	printf("%d ", h_char[i]);
	// }

    // allocate global memory
    signed char *d_char, *d_char_out, *d_char_out1, *d_charp, *d_filter;
    signed short *d_wino;
    hipMalloc( (void **) &d_char, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out1, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_charp, 34*34*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_filter, 3*3*16*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_wino, 4*4*16*16 * sizeof( signed short ) );

    hipMemcpy( d_char, h_char, SIZE * sizeof( signed char), hipMemcpyHostToDevice );
    hipMemcpy( d_filter, h_filter, 9*16*16 * sizeof( signed char), hipMemcpyHostToDevice );
    hipMemcpy( d_wino, h_wino, 16*16*16 * sizeof( signed char), hipMemcpyHostToDevice );

    signed char f;
    signed short f_short;
	FILE* fp;
    signed char x1_1[16*16*3*3];
    signed short wino[16*16*4*4];
    fp = fopen( "layer1.0.conv1.weight", "rb" );
    if (!fp) printf("x1_1: pathを間違えています\n");
    for(int i=0; i<16*16*3*3; i++){
        if( fread( &f, sizeof(f), 1, fp ) < 1 ){
            fputs( "x1_1: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        x1_1[i] = f;
    }
    hipMemcpy(d_filter, x1_1, sizeof( signed char) * 16*16*3*3, hipMemcpyHostToDevice);
    if (fp) fclose(fp);


    fp = fopen( "./wino_params_short/layer1.0.conv1.weight", "rb" );
    if (!fp) printf("wino: pathを間違えています\n");
    for(int i=0; i<16*16*4*4; i++){
        if( fread( &f_short, sizeof(f_short), 1, fp ) < 1 ){
            fputs( "wino: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        wino[i] = f_short;
    }
    hipMemcpy(d_wino, wino, sizeof(signed short) * 16*16*4*4, hipMemcpyHostToDevice);
    if (fp) fclose(fp);
    

    //Measure load store uint8
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int i=0; i<1000; i++) conv<<<16*2, 256>>>(d_char, d_filter, d_char_out);
    elapsed_time_ms=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("normal:%f\n", elapsed_time_ms);

    signed char res[32*32*16];
    hipMemcpy(res, d_char_out, sizeof( signed char) * 32*32*16, hipMemcpyDeviceToHost);
    

    //Measure load store uint8
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // make_wino<<<16, dim3(4,4,16)>>>(d_filter, f_wino);
    // padding<<<16, dim3(32,32)>>>(d_char, d_charp);
	// winograd<<<dim3(16, 16), dim3(4,4,16)>>>(d_charp, d_wino, d_char_out1);
    for(int i=0; i<1000; i++){
        padding<<<16, dim3(32,32)>>>(d_char, d_charp);
		winograd<<<dim3(16, 16), dim3(4,4,16)>>>(d_charp, d_wino, d_char_out1);
    } 
    elapsed_time_ms=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("winograd:%f\n", elapsed_time_ms);
    
    signed char res1[32*32*16];
    hipMemcpy(res1, d_char_out1, sizeof(signed char) * 32*32*16, hipMemcpyDeviceToHost);
    for(int i=0; i<32; i++){
		for(int k=0; k<32; k++) printf("%d ", char (res[k + 32*i]));
		printf("\n");
		for(int k=0; k<32; k++) printf("%d ", char (res1[k + 32*i]));
		printf("\n");
		printf("\n");
	}printf("\n");

    return;
    
}