﻿#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

#define SIZE  (16*16*32)
#define PSIZE  (18*18*32)
#define FSIZE  (3*3*32*32)
#define WSIZE  (4*4*32*32)

void initialData( signed char *a, int size){
    for (int i = 0; i < size; i++) a[i] =  i;
    return;
}

__device__  signed char clamp(int v)
{
	if(v <= -128) return -128;
	else if(v > 127) return 127;
    else return v;
}





__global__ void conv( signed char *input,  signed char *filter,  signed char *output){
    const int id = threadIdx.x, conv_ch = blockIdx.x;

	__shared__ signed char s_input[10368];  //18*18*32
	__shared__ signed char s_filter[288];   //3*3*32
    __shared__ int s_output[256];  //16*16

    // init shared memory
    for(int i = id; i < 10368; i+=blockDim.x) s_input[i] = 0;
    for(int j = id; j < 288; j+=blockDim.x) s_filter[j] = filter[j + conv_ch*288];
    for(int k = id; k < 256; k+=blockDim.x) s_output[k] = 0;

    for(int i = id; i < 8192; i+=blockDim.x){
        const int x = i&15;
        const int y = (i&255)>>4;
        const int ch = i>>8;
        s_input[(x+1) + 18*(y+1) + 324*ch] = input[i];
    }
    __syncthreads();

    for (int n = id; n < 8192; n += blockDim.x){
        const int x = n&15;
        const int y = (n&255)>>4;
        const int ch = n>>8;
        const int x0 = s_input[(x)  +18*(y)  + ch*324] * s_filter[0 + ch*9];
        const int x1 = s_input[(x+1)+18*(y)  + ch*324] * s_filter[1 + ch*9];
        const int x2 = s_input[(x+2)+18*(y)  + ch*324] * s_filter[2 + ch*9];
        const int x3 = s_input[(x)  +18*(y+1)+ ch*324] * s_filter[3 + ch*9];
        const int x4 = s_input[(x+1)+18*(y+1)+ ch*324] * s_filter[4 + ch*9];
        const int x5 = s_input[(x+2)+18*(y+1)+ ch*324] * s_filter[5 + ch*9];
        const int x6 = s_input[(x)  +18*(y+2)+ ch*324] * s_filter[6 + ch*9];
        const int x7 = s_input[(x+1)+18*(y+2)+ ch*324] * s_filter[7 + ch*9];
        const int x8 = s_input[(x+2)+18*(y+2)+ ch*324] * s_filter[8 + ch*9];
        atomicAdd(&s_output[x+(y<<4)], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }
    __syncthreads();
    for (int i = id; i < 256; i+=blockDim.x) output[i + (conv_ch<<8)] = clamp(((s_output[i] + (1 << 4)) >>5)) + 128;
}

__global__ void conv_same_tiling( signed char *input,  signed char *filter,  signed char *output){
    __shared__ int s_output[32][2][2];
    __shared__ signed char input_smem [32][4][4];
    const int tx = threadIdx.x, bx = blockIdx.x, by = blockIdx.y;
    for(int i=tx; i<512; i+=256){
        const int x = i&3, y = (i&15)>>2, z = i>>4;
        const int in_start = (bx<<1)+x + ((by<<1)+y)*18 + z*324;
        input_smem[z][y][x] = input[in_start];
    }

    if(tx < 128) {
        const int x = tx&1, y = (tx&3)>>1, z = tx>>2;
        s_output[z][y][x] = 0;
    }

    __syncthreads();
    for(int i=tx; i<4096; i+=256){  //4096 = 4*32*32
        const int x = i&1, y = (i&3)>>1, z = (i&127)>>2, ch = i>>7;
        const int f = z*9 + ch*288;
        const int x0 = input_smem[z][y+0][x+0] * filter[0 + f]; //288 = 9*32
        const int x1 = input_smem[z][y+0][x+1] * filter[1 + f];
        const int x2 = input_smem[z][y+0][x+2] * filter[2 + f];
        const int x3 = input_smem[z][y+1][x+0] * filter[3 + f];
        const int x4 = input_smem[z][y+1][x+1] * filter[4 + f];
        const int x5 = input_smem[z][y+1][x+2] * filter[5 + f];
        const int x6 = input_smem[z][y+2][x+0] * filter[6 + f];
        const int x7 = input_smem[z][y+2][x+1] * filter[7 + f];
        const int x8 = input_smem[z][y+2][x+2] * filter[8 + f];
        atomicAdd(&s_output[ch][y][x], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }
    
    __syncthreads();
    if(tx < 128){
        const int x = tx&1, y = (tx&3)>>1, z = tx>>2;
        const int out_start = (bx<<1)+x+1 + ((by<<1)+y+1)*18 + (z*324); 
        output[out_start] = clamp(((s_output[z][y][x] + (1 << 4)) >>5)) + 128;   
    }
}


__global__ void winograd( signed char *input,  signed short *weight,  signed char  *output){
	// dim3(32/2, 32/2) dim3(4,4,16)
    const int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z, bx = blockIdx.x, by = blockIdx.y;
	const int in_start_xy = (bx<<1) + tx + ((by<<1)+ty)*18;  //324 = 18*18
    const int x_y = tx + (ty<<2);
    const int id = x_y + (threadIdx.z<<4);
	__shared__ signed char input_smem [32][16];
	__shared__ int BtdB [32][16];
	__shared__ int I [32][4][4];
	
	// I[tz][ty][tx] = 0;
	// input_smem[tz][x_y] = input[in_start];
    for(int i=0; i < 2; i++){
        const int in_start = in_start_xy + (tz+(i<<4))*324;
	    input_smem[tz+(i<<4)][x_y] = input[in_start];
	    I[tz+(i<<4)][ty][tx] = 0;
    }
    __syncthreads();
    if(id < 32){
        BtdB[id][0] = input_smem[id][0]-input_smem[id][8]-input_smem[id][2]+input_smem[id][10];
        BtdB[id][1] = input_smem[id][1]-input_smem[id][9]+input_smem[id][2]-input_smem[id][10];
        BtdB[id][2] = -input_smem[id][1]+input_smem[id][9]+input_smem[id][2]-input_smem[id][10];
        BtdB[id][3] = input_smem[id][1]-input_smem[id][9]-input_smem[id][3]+input_smem[id][11];
        BtdB[id][4] = input_smem[id][4]+input_smem[id][8]-input_smem[id][6]-input_smem[id][10];
        BtdB[id][5] = input_smem[id][5]+input_smem[id][9]+input_smem[id][6]+input_smem[id][10];
        BtdB[id][6] = -input_smem[id][5]-input_smem[id][9]+input_smem[id][6]+input_smem[id][10];
        BtdB[id][7] = input_smem[id][5]+input_smem[id][9]-input_smem[id][7]-input_smem[id][11];
        BtdB[id][8] = -input_smem[id][4]+input_smem[id][8]+input_smem[id][6]-input_smem[id][10];
        BtdB[id][9] = -input_smem[id][5]+input_smem[id][9]-input_smem[id][6]+input_smem[id][10];
        BtdB[id][10] = input_smem[id][5]-input_smem[id][9]-input_smem[id][6]+input_smem[id][10];
        BtdB[id][11] = -input_smem[id][5]+input_smem[id][9]+input_smem[id][7]-input_smem[id][11];
        BtdB[id][12] = input_smem[id][4]-input_smem[id][12]-input_smem[id][6]+input_smem[id][14];
        BtdB[id][13] = input_smem[id][5]-input_smem[id][13]+input_smem[id][6]-input_smem[id][14];
        BtdB[id][14] = -input_smem[id][5]+input_smem[id][13]+input_smem[id][6]-input_smem[id][14];
        BtdB[id][15] = input_smem[id][5]-input_smem[id][13]-input_smem[id][7]+input_smem[id][15];
    }
    __syncthreads();
    for(int i=id; i<16384; i+=256){ //16384 = 4*4*32*32
        const int ch = i>>9;
		atomicAdd(&I[ch][ty][tx], BtdB[tz][x_y]*weight[i]);
	}
	__syncthreads();
    if(id < 32) {
        const int out_start1 = ((bx<<1)+1) + (((by<<1)+1)*18) + ((id)*324);
        const int out_start2 = ((bx<<1)+2) + (((by<<1)+1)*18) + ((id)*324);
        const int out_start3 = ((bx<<1)+1) + (((by<<1)+2)*18) + ((id)*324);
        const int out_start4 = ((bx<<1)+2) + (((by<<1)+2)*18) + ((id)*324);
        output[out_start1] = clamp((((I[id][0][0] + I[id][0][1] + I[id][0][2] + I[id][1][0] + I[id][1][1] + I[id][1][2] + I[id][2][0] + I[id][2][1] + I[id][2][2]) + (1 << 6)) >>7)) + 128;
        output[out_start2] = clamp((((I[id][0][1] - I[id][0][2] - I[id][0][3] + I[id][1][1] - I[id][1][2] - I[id][1][3] + I[id][2][1] - I[id][2][2] - I[id][2][3]) + (1 << 6)) >>7)) + 128;
        output[out_start3] = clamp((((I[id][1][0] + I[id][1][1] + I[id][1][2] - I[id][2][0] - I[id][2][1] - I[id][2][2] - I[id][3][0] - I[id][3][1] - I[id][3][2]) + (1 << 6)) >>7)) + 128;
        output[out_start4] = clamp((((I[id][1][1] - I[id][1][2] - I[id][1][3] - I[id][2][1] + I[id][2][2] + I[id][2][3] - I[id][3][1] + I[id][3][2] + I[id][3][3]) + (1 << 6)) >>7)) + 128;
    }
}


__global__ void padding( signed char *input,  signed char *output){
    const int id  = threadIdx.x + blockDim.x*threadIdx.y;
    const int idx = threadIdx.x;
    const int idy = threadIdx.y;
    const int ch = blockIdx.x;
    __shared__  signed char s_output[18*18];

    for(int i=id; i< 18*18; i+=blockDim.x*blockDim.y) s_output[i] = 0;
    __syncthreads();
    s_output[(idx+1) + (idy+1)*18] = input[idx + (idy<<4) + (ch<<8)];
    __syncthreads();
    for(int i=id; i< 18*18; i+=blockDim.x*blockDim.y) output[i + ch*18*18] = s_output[i];
}


int main(){
    hipEvent_t start, stop;
    float elapsed_time_ms1, elapsed_time_ms2, elapsed_time_ms3;
    signed char *h_char = ( signed char *)malloc(SIZE * sizeof( signed char));

    initialData(h_char, SIZE);
    // allocate global memory
    signed char *d_char, *d_char_out, *d_char_outp, *d_charp, *d_filter;
    signed short *d_wino;
    hipMalloc( (void **) &d_char, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_outp, PSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_charp, PSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_filter, FSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_wino, WSIZE * sizeof( signed short) );

    hipMemcpy( d_char, h_char, SIZE * sizeof( signed char), hipMemcpyHostToDevice );

    signed char f;
    signed short f_short;
	FILE* fp;
    signed char x1_1[FSIZE];
    signed short wino[WSIZE];
    fp = fopen( "./params/layer2.0.conv2.weight", "rb" );
    if (!fp) printf("x1_1: pathを間違えています\n");
    for(int i=0; i<FSIZE; i++){
        if( fread( &f, sizeof(f), 1, fp ) < 1 ){
            fputs( "x1_1: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        x1_1[i] = f;
    }
    hipMemcpy(d_filter, x1_1, sizeof(signed char) * FSIZE, hipMemcpyHostToDevice);
    if (fp) fclose(fp);


    fp = fopen( "./wino_params_short/layer2.0.conv2.weight", "rb" );
    if (!fp) printf("wino: pathを間違えています\n");
    for(int i=0; i<WSIZE; i++){
        if( fread( &f_short, sizeof(f_short), 1, fp ) < 1 ){
            fputs( "wino: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        wino[i] = f_short;
    }
    hipMemcpy(d_wino, wino, sizeof(signed short) * WSIZE, hipMemcpyHostToDevice);
    if (fp) fclose(fp);
    
    
    //Measure
    padding<<<32, dim3(16,16)>>>(d_char, d_charp);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMemset(&d_char_outp, 0, sizeof(signed char)*PSIZE);
    winograd<<<dim3(8, 8), dim3(4,4,16)>>>(d_charp, d_wino, d_char_outp);
    elapsed_time_ms2=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms2, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("winograd:%f\n", elapsed_time_ms2);
    signed char res1[PSIZE];
    hipMemcpy(res1, d_char_outp, sizeof(signed char) * PSIZE, hipMemcpyDeviceToHost);
    

    //Measure
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    conv<<<32, 256>>>(d_char, d_filter, d_char_out);
    elapsed_time_ms1=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("normal:%f\n", elapsed_time_ms1);

    //Measure
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    conv_same_tiling<<<dim3(8, 8), 256>>>(d_charp, d_filter, d_char_outp);
    elapsed_time_ms3=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms3, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("same_tiling:%f\n", elapsed_time_ms3);
    signed char res2[PSIZE];
    hipMemcpy(res2, d_char_outp, sizeof(signed char) * PSIZE, hipMemcpyDeviceToHost);


    
    //check result
    signed char res[SIZE];
    hipMemcpy(res, d_char_out, sizeof(signed char) * SIZE, hipMemcpyDeviceToHost);

    signed char resp[PSIZE] = {0};
    for(int i=0;i<32;i++){
        for (int j=0;j<16; j++){
            for (int k=0;k<16; k++){
                resp[j+1 + (k+1)*18 + i*324] = res[j + k*16 + i*256];
            }
        }
    }

    int miss = 0;
    for(int i=0;i<PSIZE; i++) if(resp[i] != res1[i] || resp[i] != res2[i]) {miss++;}
    if(miss == 0) printf("%f 倍速くなりました。normal/wino\n", elapsed_time_ms1/elapsed_time_ms2);
    if(miss == 0) printf("%f 倍速くなりました。same_tiling/wino\n", elapsed_time_ms3/elapsed_time_ms2);
    if(miss == 0) printf("%f 倍速くなりました。normal/same_tiling\n", elapsed_time_ms1/elapsed_time_ms3);
    else if(miss != 0) printf("miss = %d bat!", miss);

    free(h_char );
    hipFree(d_char);
    hipFree(d_char_out);
    hipFree(d_char_outp);
    hipFree(d_charp);
    hipFree(d_filter);
    hipFree(d_wino);

    return 0;
    
}