#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include <iostream>
#include <random>

#define SIZE  (32*32*16)
#define PSIZE  (34*34*16)

void initialData( signed char *a, int size){
    for (int i = 0; i < size; i++) a[i] =  signed char(i);
    return;
}

__device__  signed char clamp(int v)
{
	if(v <= -128) return -128;
	else if(v > 127) return 127;
    else return v;
}


__global__ void conv( signed char *input,  signed char *filter,  signed char *output){
    __shared__ int s_output[16];
    __shared__ signed char input_smem [16][9];
    const int tz = threadIdx.x, bx = blockIdx.x, by = blockIdx.y;
    for(int i=0; i<9; i++){
        const int x = i%3, y = i/3;
        const int in_start = bx+x + (by+y)*34 + tz*1156;  //1156 = 34*34
        input_smem[tz][i] = input[in_start];
    }
    
    s_output[tz] = 0;
    __syncthreads();
    for(int i=0; i<16; i++){
        const int x0 = input_smem[tz][0] * filter[0 + tz*9 + i*144]; //144 = 9*16
        const int x1 = input_smem[tz][1] * filter[1 + tz*9 + i*144];
        const int x2 = input_smem[tz][2] * filter[2 + tz*9 + i*144];
        const int x3 = input_smem[tz][3] * filter[3 + tz*9 + i*144];
        const int x4 = input_smem[tz][4] * filter[4 + tz*9 + i*144];
        const int x5 = input_smem[tz][5] * filter[5 + tz*9 + i*144];
        const int x6 = input_smem[tz][6] * filter[6 + tz*9 + i*144];
        const int x7 = input_smem[tz][7] * filter[7 + tz*9 + i*144];
        const int x8 = input_smem[tz][8] * filter[8 + tz*9 + i*144];
        atomicAdd(&s_output[i], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }
    __syncthreads();
    const int out_start = bx+1 + (by+1)*34 + (tz*1156);
    output[out_start] = clamp(((s_output[tz] + (1 << 4)) >>5)) + 128;
}


__global__ void winograd( signed char *input,  signed short *weight,  signed char *output){
	// dim3(32/2, 32/2) dim3(4,4,16)
    const int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z, bx = blockIdx.x, by = blockIdx.y;
	const int in_start = bx*2 + tx + (by*2+ty)*34 + tz*1156;  //1156 = 34*34

	// dim3(32/2, 32/2, 16) dim3(16,4,4)
	// const int in_start = tx + ((ty + (bx<<1))<<4) + (tz + (by<<1))*544;  //1156 = 34*34


	__shared__ signed char input_smem [16][4][4];
	__shared__ int Btd [16][4][4];
	__shared__ int BtdB [16][4][4];
	__shared__ int I [16][4][4];
	
	I[tz][ty][tx] = 0;
	input_smem[tz][ty][tx] = input[in_start];
	// __syncthreads();
	switch (ty)
	{
	case 0:
		Btd [tz][ty][tx] = input_smem[tz][tx][0] - input_smem[tz][tx][2];
		break;
	case 1:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 2:
		Btd [tz][ty][tx] = - input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 3:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] - input_smem[tz][tx][3];
		break;
	}
	// __syncthreads();
	switch (tx)
	{
	case 0:
		BtdB[tz][tx][ty] = Btd[tz][ty][0] - Btd[tz][ty][2];
		break;
	case 1:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 2:
		BtdB[tz][tx][ty] = - Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 3:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] - Btd[tz][ty][3];
		break;
	}
	// __syncthreads();
    const int id = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
	for(int i=id; i<4*4*16*16; i+=blockDim.x*blockDim.y*blockDim.z){
        const int ch = i>>8;
		atomicAdd(&I[ch][ty][tx], BtdB[tz][ty][tx]*weight[i]);
	}
    __syncthreads();
    if(id < 16) {
        const int out_start1 = (bx*2+1) + ((by*2+1)*34) + ((id)*1156);
        const int out_start2 = (bx*2+2) + ((by*2+1)*34) + ((id)*1156);
        const int out_start3 = (bx*2+1) + ((by*2+2)*34) + ((id)*1156);
        const int out_start4 = (bx*2+2) + ((by*2+2)*34) + ((id)*1156);
        output[out_start1] = clamp((((I[id][0][0] + I[id][0][1] + I[id][0][2] + I[id][1][0] + I[id][1][1] + I[id][1][2] + I[id][2][0] + I[id][2][1] + I[id][2][2]) + (1 << 6)) >>7)) + 128;
        output[out_start2] = clamp((((I[id][0][1] - I[id][0][2] - I[id][0][3] + I[id][1][1] - I[id][1][2] - I[id][1][3] + I[id][2][1] - I[id][2][2] - I[id][2][3]) + (1 << 6)) >>7)) + 128;
        output[out_start3] = clamp((((I[id][1][0] + I[id][1][1] + I[id][1][2] - I[id][2][0] - I[id][2][1] - I[id][2][2] - I[id][3][0] - I[id][3][1] - I[id][3][2]) + (1 << 6)) >>7)) + 128;
        output[out_start4] = clamp((((I[id][1][1] - I[id][1][2] - I[id][1][3] - I[id][2][1] + I[id][2][2] + I[id][2][3] - I[id][3][1] + I[id][3][2] + I[id][3][3]) + (1 << 6)) >>7)) + 128;
    }
}

__global__ void padding( signed char *input,  signed char *output){
    const int id  = threadIdx.x + blockDim.x*threadIdx.y;
    const int idx = threadIdx.x;
    const int idy = threadIdx.y;
    const int ch = blockIdx.x;
    __shared__  signed char s_output[34*34];

    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) s_output[i] = 0;
    __syncthreads();
    s_output[(idx+1) + (idy+1)*34] = input[idx + idy*32 + ch*32*32];
    __syncthreads();
    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) output[i + ch*34*34] = s_output[i];
}


int main(){
    hipEvent_t start, stop;
    float elapsed_time_ms1, elapsed_time_ms2;
    signed char *h_char = ( signed char *)malloc(SIZE * sizeof( signed char));
    initialData(h_char, SIZE);

    // allocate global memory
    signed char *d_char, *d_char_out, *d_char_outp, *d_charp, *d_filter;
    signed short *d_wino;
    hipMalloc( (void **) &d_char, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_outp, 34*34*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_charp, 34*34*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_filter, 3*3*16*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_wino, 4*4*16*16 * sizeof( signed short ) );

    hipMemcpy( d_char, h_char, SIZE * sizeof( signed char), hipMemcpyHostToDevice );

    signed char f;
    signed short f_short;
	FILE* fp;
    signed char x1_1[16*16*3*3];
    signed short wino[16*16*4*4];
    fp = fopen( "./params/layer1.0.conv1.weight", "rb" );
    if (!fp) printf("x1_1: pathを間違えています\n");
    for(int i=0; i<16*16*3*3; i++){
        if( fread( &f, sizeof(f), 1, fp ) < 1 ){
            fputs( "x1_1: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        x1_1[i] = f;
    }
    hipMemcpy(d_filter, x1_1, sizeof( signed char) * 16*16*3*3, hipMemcpyHostToDevice);
    if (fp) fclose(fp);


    fp = fopen( "./wino_params_short/layer1.0.conv1.weight", "rb" );
    if (!fp) printf("wino: pathを間違えています\n");
    for(int i=0; i<16*16*4*4; i++){
        if( fread( &f_short, sizeof(f_short), 1, fp ) < 1 ){
            fputs( "wino: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        wino[i] = f_short;
    }
    hipMemcpy(d_wino, wino, sizeof(signed short) * 16*16*4*4, hipMemcpyHostToDevice);
    if (fp) fclose(fp);
    

    padding<<<16, dim3(32,32)>>>(d_char, d_charp);
    //Measure load store uint8
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int i=0; i<1000; i++) {
        hipMemset(&d_char_outp, 0, sizeof(signed char)*PSIZE);
        conv<<<dim3(32, 32), dim3(16)>>>(d_charp, d_filter, d_char_outp);
    }
    elapsed_time_ms1=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("normal:%f\n", elapsed_time_ms1);

    signed char res[PSIZE];
    hipMemcpy(res, d_char_outp, sizeof(signed char) * PSIZE, hipMemcpyDeviceToHost);
    

    //Measure load store uint8
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int i=0; i<1000; i++){
        hipMemset(&d_char_outp, 0, sizeof(signed char)*(34*34*16));
		winograd<<<dim3(16, 16), dim3(4,4,16)>>>(d_charp, d_wino, d_char_outp);
    } 
    elapsed_time_ms2=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms2, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("winograd:%f\n", elapsed_time_ms2);
    
    signed char res1[34*34*16];
    hipMemcpy(res1, d_char_outp, sizeof(signed char) * 34*34*16, hipMemcpyDeviceToHost);

    //check data
    int miss = 0;
    for(int i=0;i<PSIZE; i++) if(res[i] != res1[i]) {printf("%d ", i); miss++;}
    if(miss == 0) printf("%f 倍速くなりました。", elapsed_time_ms1/elapsed_time_ms2);
    else if(miss != 0) printf("%f 倍速くなりました。答え一致してないけどね", elapsed_time_ms1/elapsed_time_ms2);
    return;

    return;
    
}