#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include <iostream>
#include <random>

#define SIZE  (16*16*32)
#define OUTSIZE  (8*8*64)
#define PSIZE  (18*18*32)
#define POUTSIZE  (10*10*64)
#define FSIZE  (3*3*32*64)
#define WSIZE  (4*4*32*64)

void initialData( signed char *a, int size){
    for (int i = 0; i < size; i++) a[i] =  signed char(i);
    return;
}

__device__  signed char clamp(int v)
{
	if(v <= -128) return -128;
	else if(v > 127) return 127;
    else return v;
}

__global__ void conv( signed char *input,  signed char *filter,  signed char *output){
    // input = (16,16,32), output =(8,8,64), filter =(3,3,32,64), s_input = (18,18,32)
    
    const int id = threadIdx.x, conv_ch = blockIdx.x;

	__shared__ signed char s_input[10368];  //18*18*32
	__shared__ signed char s_filter[288];   //3*3*32
    __shared__ int s_output[64];  //8*8

    // init shared memory
    for(int i = id; i < 10368; i+=blockDim.x) s_input[i] = 0;
    for(int j = id; j < 288; j+=blockDim.x) s_filter[j] = filter[j + conv_ch*288];
    for(int k = id; k < 64; k+=blockDim.x) s_output[k] = 0;

    for(int i = id; i < 8192; i+=blockDim.x){
        const int x = i&15;
        const int y = (i&255)>>4;
        const int ch = i>>8;
        s_input[(x+1) + 18*(y+1) + 324*ch] = input[i];
    }
    __syncthreads();
    for (int n = id; n < 2048; n += blockDim.x){   //4096 = output_2d * input_ch
        const int x = n&7;
        const int y = (n&63)>>3;
        const int ch = n>>6;
        const int x0 = s_input[((x<<1))  +18*((y<<1))  + ch*324] * s_filter[0 + ch*9];
        const int x1 = s_input[((x<<1)+1)+18*((y<<1))  + ch*324] * s_filter[1 + ch*9];
        const int x2 = s_input[((x<<1)+2)+18*((y<<1))  + ch*324] * s_filter[2 + ch*9];
        const int x3 = s_input[((x<<1))  +18*((y<<1)+1)+ ch*324] * s_filter[3 + ch*9];
        const int x4 = s_input[((x<<1)+1)+18*((y<<1)+1)+ ch*324] * s_filter[4 + ch*9];
        const int x5 = s_input[((x<<1)+2)+18*((y<<1)+1)+ ch*324] * s_filter[5 + ch*9];
        const int x6 = s_input[((x<<1))  +18*((y<<1)+2)+ ch*324] * s_filter[6 + ch*9];
        const int x7 = s_input[((x<<1)+1)+18*((y<<1)+2)+ ch*324] * s_filter[7 + ch*9];
        const int x8 = s_input[((x<<1)+2)+18*((y<<1)+2)+ ch*324] * s_filter[8 + ch*9];
        atomicAdd(&s_output[x+(y<<3)], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }
    __syncthreads();
    for (int i = id; i < 64; i+=blockDim.x) output[i + (conv_ch<<6)] = clamp(((s_output[i] + (1 << 4)) >>5)) + 128;
}


__global__ void winograd( signed char *input,  signed short *weight,  signed char  *output){
	// dim3(8/2, 8/2) dim3(4,4,64)
    const int id = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    const int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z, bx = blockIdx.x, by = blockIdx.y;
	const int in_start = bx*2 + tx + (by*2+ty)*18 + tz*324;  //324 = 18*18
	
    

	__shared__ signed char input_smem [32][4][4];
	__shared__ int output_smem [64][2][2];
	__shared__ int Btd [32][4][4];
	__shared__ int BtdB [32][4][4];
	__shared__ int AtI [64][2][4];
	__shared__ int I [64][4][4];
	
	I[tz][ty][tx] = 0;
	input_smem[tz][ty][tx] = input[in_start];
	// __syncthreads();
	switch (ty)
	{
	case 0:
		Btd [tz][ty][tx] = input_smem[tz][tx][0] - input_smem[tz][tx][2];
		break;
	case 1:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 2:
		Btd [tz][ty][tx] = - input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 3:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] - input_smem[tz][tx][3];
		break;
	}
	// __syncthreads();
	switch (tx)
	{
	case 0:
		BtdB[tz][tx][ty] = Btd[tz][ty][0] - Btd[tz][ty][2];
		break;
	case 1:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 2:
		BtdB[tz][tx][ty] = - Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 3:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] - Btd[tz][ty][3];
		break;
	}
	__syncthreads();
	for(int i=id; i<512*64; i+=512){
        const int ch = i>>9;
		atomicAdd(&I[ch][ty][tx], BtdB[tz][ty][tx]*weight[i]);
	}

    // __syncthreads();
    if(tx == 0 && ty == 0) {
        const int out_start1 = (bx+1) + ((by+1)*10) + ((2*tz)*100);
        const int out_start2 = (bx+1) + ((by+1)*10) + ((2*tz + 1)*100);
        output[out_start1] = clamp((((I[2*tz][0][0] + I[2*tz][0][1] + I[2*tz][0][2] + I[2*tz][1][0] + I[2*tz][1][1] + I[2*tz][1][2] + I[2*tz][2][0] + I[2*tz][2][1] + I[2*tz][2][2]) + (1 << 6)) >>7)) + 128;
        output[out_start2] = clamp((((I[2*tz + 1][0][0] + I[2*tz + 1][0][1] + I[2*tz + 1][0][2] + I[2*tz + 1][1][0] + I[2*tz + 1][1][1] + I[2*tz + 1][1][2] + I[2*tz + 1][2][0] + I[2*tz + 1][2][1] + I[2*tz + 1][2][2]) + (1 << 6)) >>7)) + 128;
    }
	
}


__global__ void padding( signed char *input,  signed char *output){
    const int id  = threadIdx.x + blockDim.x*threadIdx.y;
    const int idx = threadIdx.x;
    const int idy = threadIdx.y;
    const int ch = blockIdx.x;
    __shared__  signed char s_output[18*18];

    for(int i=id; i< 18*18; i+=blockDim.x*blockDim.y) s_output[i] = 0;
    __syncthreads();
    s_output[(idx+1) + (idy+1)*18] = input[idx + (idy<<4) + (ch<<8)];
    __syncthreads();
    for(int i=id; i< 18*18; i+=blockDim.x*blockDim.y) output[i + ch*18*18] = s_output[i];
}


int main(){
    hipEvent_t start, stop;
    float elapsed_time_ms;
     signed char *h_char = ( signed char *)malloc(SIZE * sizeof( signed char));

    initialData(h_char, SIZE);

    // allocate global memory
    signed char *d_char, *d_char_out, *d_char_outp, *d_charp, *d_filter;
    signed short *d_wino;
    hipMalloc( (void **) &d_char, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out, OUTSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_outp, POUTSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_charp, PSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_filter, FSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_wino, WSIZE * sizeof( signed short) );

    hipMemcpy( d_char, h_char, SIZE * sizeof( signed char), hipMemcpyHostToDevice );

    signed char f;
    signed short f_short;
	FILE* fp;
    signed char x1_1[FSIZE];
    signed short wino[WSIZE];
    fp = fopen( "./layer3.0.conv1.weight", "rb" );
    if (!fp) printf("x1_1: pathを間違えています\n");
    for(int i=0; i<FSIZE; i++){
        if( fread( &f, sizeof(f), 1, fp ) < 1 ){
            fputs( "x1_1: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        x1_1[i] = f;
    }
    hipMemcpy(d_filter, x1_1, sizeof( signed char) * FSIZE, hipMemcpyHostToDevice);
    if (fp) fclose(fp);


    fp = fopen( "./wino_params_short/layer3.0.conv1.weight", "rb" );
    if (!fp) printf("wino: pathを間違えています\n");
    for(int i=0; i<WSIZE; i++){
        if( fread( &f_short, sizeof(f_short), 1, fp ) < 1 ){
            fputs( "wino: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        wino[i] = f_short;
    }
    hipMemcpy(d_wino, wino, sizeof(signed short) * WSIZE, hipMemcpyHostToDevice);
    if (fp) fclose(fp);
    
    
    //Measure load store uint8
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int i=0; i<1000; i++) {
        // hipMemset(&d_char_outp, 0, sizeof(signed char)*PSIZE);
        conv<<<64, 256>>>(d_char, d_filter, d_char_out);
    }
    elapsed_time_ms=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("normal:%f\n", elapsed_time_ms);

    signed char res[OUTSIZE];
    hipMemcpy(res, d_char_out, sizeof(signed char) * OUTSIZE, hipMemcpyDeviceToHost);
    for(int i=0; i<32; i++) printf("%d, ", res[i]);
    printf("\n");

    //Measure load store uint8
    padding<<<32, dim3(16,16)>>>(d_char, d_charp);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int i=0; i<1000; i++){
        hipMemset(&d_char_outp, 0, sizeof(signed char)*PSIZE);
		winograd<<<dim3(8, 8), dim3(4,4,32)>>>(d_charp, d_wino, d_char_outp);
    } 
    elapsed_time_ms=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("winograd:%f\n", elapsed_time_ms);
    
    signed char res1[POUTSIZE];
    hipMemcpy(res1, d_char_outp, sizeof(signed char) * POUTSIZE, hipMemcpyDeviceToHost);
    for(int i=11; i<32; i++) printf("%d, ", char (res1[i]));
    printf("\n");


    free(h_char );
    hipFree(d_char);
    hipFree(d_char_out);
    hipFree(d_char_outp);
    hipFree(d_charp);
    hipFree(d_filter);
    hipFree(d_wino);

    return;
    
}