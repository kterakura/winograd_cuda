﻿#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <random>

#define SIZE  (32*32*16)
#define PSIZE  (34*34*16)

void initialData( signed char *a, int size){
    for (int i = 0; i < size; i++) a[i] =  i;
    return;
}

__device__  signed char clamp(int v)
{
	if(v <= -128) return -128;
	else if(v > 127) return 127;
    else return v;
}


__global__ void conv( signed char *input,  signed char *filter,  signed char *output){
    // input = (32,16,16), output =(32,16,16), filter =(3,3,16,16), s_input = (34,18,16)

    const int id = threadIdx.x + blockDim.x*threadIdx.y, ch_id = blockIdx.x, block_size = blockDim.x*blockDim.y;
    const int conv_place = (ch_id&1), conv_ch = (ch_id>>1);
    
    __shared__  signed char s_input[9792]; //34*18*16
	__shared__  signed char s_filter[144]; //3*3*16
    __shared__ int s_output[512];  //32*16
    
    // init shared memory
    for(int i = id; i < 9792; i+=block_size) s_input[i] = 0;
    for(int j = id; j < 144; j+=block_size) s_filter[j] = filter[j + conv_ch*144];
    for(int k = id; k < 512; k+=block_size) s_output[k] = 0;

    for (int i = id; i < 8704; i+=blockDim.x){   //8704 = 32*(16+1)*16,   544 = 32*(16+1)
        const int x = i&31, y = ((i%544)>>5), ch = i/(544);
        s_input[(x+1) + 34*(y+(conv_place^1)) + 612*ch] = input[x + ((y + (conv_place<<4) - conv_place)<<5) + (ch<<10)];
    }__syncthreads();

    for (int n = id; n < 8192; n += block_size){  //9782 = output_2d * input_ch
        const int x = n&31;
        const int y = (n&511)>>5;
        const int ch = n>>9;
        const int x0 = s_input[(x)  +34*(y)  + 612*ch] * s_filter[0 + 9*ch];
        const int x1 = s_input[(x+1)+34*(y)  + 612*ch] * s_filter[1 + 9*ch];
        const int x2 = s_input[(x+2)+34*(y)  + 612*ch] * s_filter[2 + 9*ch];
        const int x3 = s_input[(x)  +34*(y+1)+ 612*ch] * s_filter[3 + 9*ch];
        const int x4 = s_input[(x+1)+34*(y+1)+ 612*ch] * s_filter[4 + 9*ch];
        const int x5 = s_input[(x+2)+34*(y+1)+ 612*ch] * s_filter[5 + 9*ch];
        const int x6 = s_input[(x)  +34*(y+2)+ 612*ch] * s_filter[6 + 9*ch];
        const int x7 = s_input[(x+1)+34*(y+2)+ 612*ch] * s_filter[7 + 9*ch];
        const int x8 = s_input[(x+2)+34*(y+2)+ 612*ch] * s_filter[8 + 9*ch];
        atomicAdd(&s_output[x+(y<<5)], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }

    __syncthreads();
    for (int i = id; i < 512; i+=blockDim.x){
        const int x = i&31, y = (i>>5) + (conv_place<<4);
        output[x + (y<<5) + (conv_ch<<10)] = clamp(((s_output[i] + (1 << 4)) >>5)) + 128;
    }
}


__global__ void winograd( signed char *input,  signed short *weight,  signed char *output){
	// dim3(32/2, 32/2) dim3(4,4,16)
    const int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z, bx = blockIdx.x, by = blockIdx.y;
	const int in_start = bx*2 + tx + (by*2+ty)*34 + tz*1156;  //1156 = 34*34

	// dim3(32/2, 32/2, 16) dim3(16,4,4)
	// const int in_start = tx + ((ty + (bx<<1))<<4) + (tz + (by<<1))*544;  //1156 = 34*34


	__shared__ signed char input_smem [16][4][4];
	__shared__ int Btd [16][4][4];
	__shared__ int BtdB [16][4][4];
	__shared__ int I [16][4][4];
	
	I[tz][ty][tx] = 0;
	input_smem[tz][ty][tx] = input[in_start];
	// __syncthreads();
	switch (ty)
	{
	case 0:
		Btd [tz][ty][tx] = input_smem[tz][tx][0] - input_smem[tz][tx][2];
		break;
	case 1:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 2:
		Btd [tz][ty][tx] = - input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 3:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] - input_smem[tz][tx][3];
		break;
	}
	// __syncthreads();
	switch (tx)
	{
	case 0:
		BtdB[tz][tx][ty] = Btd[tz][ty][0] - Btd[tz][ty][2];
		break;
	case 1:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 2:
		BtdB[tz][tx][ty] = - Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 3:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] - Btd[tz][ty][3];
		break;
	}
	// __syncthreads();
    const int id = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
	for(int i=id; i<4*4*16*16; i+=blockDim.x*blockDim.y*blockDim.z){
        const int ch = i>>8;
		atomicAdd(&I[ch][ty][tx], BtdB[tz][ty][tx]*weight[i]);
	}
    __syncthreads();
    if(id < 16) {
        const int out_start1 = (bx*2+1) + ((by*2+1)*34) + ((id)*1156);
        const int out_start2 = (bx*2+2) + ((by*2+1)*34) + ((id)*1156);
        const int out_start3 = (bx*2+1) + ((by*2+2)*34) + ((id)*1156);
        const int out_start4 = (bx*2+2) + ((by*2+2)*34) + ((id)*1156);
        output[out_start1] = clamp((((I[id][0][0] + I[id][0][1] + I[id][0][2] + I[id][1][0] + I[id][1][1] + I[id][1][2] + I[id][2][0] + I[id][2][1] + I[id][2][2]) + (1 << 6)) >>7)) + 128;
        output[out_start2] = clamp((((I[id][0][1] - I[id][0][2] - I[id][0][3] + I[id][1][1] - I[id][1][2] - I[id][1][3] + I[id][2][1] - I[id][2][2] - I[id][2][3]) + (1 << 6)) >>7)) + 128;
        output[out_start3] = clamp((((I[id][1][0] + I[id][1][1] + I[id][1][2] - I[id][2][0] - I[id][2][1] - I[id][2][2] - I[id][3][0] - I[id][3][1] - I[id][3][2]) + (1 << 6)) >>7)) + 128;
        output[out_start4] = clamp((((I[id][1][1] - I[id][1][2] - I[id][1][3] - I[id][2][1] + I[id][2][2] + I[id][2][3] - I[id][3][1] + I[id][3][2] + I[id][3][3]) + (1 << 6)) >>7)) + 128;
    }

	// if(ty > 1) return;
	// switch (ty)
	// {
	// case 0:
	// 	AtI[tz][ty][tx] = I[tz][0][tx] + I[tz][1][tx] + I[tz][2][tx];
	// 	break;
	// case 1:
	// 	AtI[tz][ty][tx] = I[tz][1][tx] - I[tz][2][tx] - I[tz][3][tx];
	// 	break;
	// }
	// // __syncthreads();

	// if(tx > 1) return;
	// switch (tx)
	// {
	// case 0:
	// 	output[out_start] = clamp((((AtI[tz][ty][0] + AtI[tz][ty][1] + AtI[tz][ty][2]) + (1 << 6)) >>7)) + 128;
	// 	break;
	// case 1:
	// 	output[out_start] = clamp((((AtI[tz][ty][1] - AtI[tz][ty][2] - AtI[tz][ty][3]) + (1 << 6)) >>7)) + 128;
	// 	break;
	// }
	// __syncthreads();
	// output[out_start] = clamp(((output_smem[tz][ty][tx] + (1 << 4)) >>5)) + 128;
}

__global__ void padding( signed char *input,  signed char *output){
    const int id  = threadIdx.x + blockDim.x*threadIdx.y;
    const int idx = threadIdx.x;
    const int idy = threadIdx.y;
    const int ch = blockIdx.x;
    __shared__  signed char s_output[34*34];

    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) s_output[i] = 0;
    __syncthreads();
    s_output[(idx+1) + (idy+1)*34] = input[idx + idy*32 + ch*32*32];
    __syncthreads();
    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) output[i + ch*34*34] = s_output[i];
}


int main(){
    hipEvent_t start, stop;
    float elapsed_time_ms1, elapsed_time_ms2;
    signed char *h_char = ( signed char *)malloc(SIZE * sizeof( signed char));
    initialData(h_char, SIZE);

    // allocate global memory
    signed char *d_char, *d_char_out, *d_char_outp, *d_charp, *d_filter;
    signed short *d_wino;
    hipMalloc( (void **) &d_char, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_outp, 34*34*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_charp, 34*34*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_filter, 3*3*16*16 * sizeof( signed char) );
    hipMalloc( (void **) &d_wino, 4*4*16*16 * sizeof( signed short ) );

    hipMemcpy( d_char, h_char, SIZE * sizeof( signed char), hipMemcpyHostToDevice );

    signed char f;
    signed short f_short;
	FILE* fp;
    signed char x1_1[16*16*3*3];
    signed short wino[16*16*4*4];
    fp = fopen( "./params/layer1.0.conv1.weight", "rb" );
    if (!fp) printf("x1_1: pathを間違えています\n");
    for(int i=0; i<16*16*3*3; i++){
        if( fread( &f, sizeof(f), 1, fp ) < 1 ){
            fputs( "x1_1: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        x1_1[i] = f;
    }
    hipMemcpy(d_filter, x1_1, sizeof( signed char) * 16*16*3*3, hipMemcpyHostToDevice);
    if (fp) fclose(fp);


    fp = fopen( "./wino_params_short/layer1.0.conv1.weight", "rb" );
    if (!fp) printf("wino: pathを間違えています\n");
    for(int i=0; i<16*16*4*4; i++){
        if( fread( &f_short, sizeof(f_short), 1, fp ) < 1 ){
            fputs( "wino: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        wino[i] = f_short;
    }
    hipMemcpy(d_wino, wino, sizeof(signed short) * 16*16*4*4, hipMemcpyHostToDevice);
    if (fp) fclose(fp);
    

    //Measure
    float time1 = 0;
    for(int i=0; i<1000; i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        conv<<<16*2, 256>>>(d_char, d_filter, d_char_out);
        elapsed_time_ms1=0.0f;
        hipEventRecord(stop, 0);
        hipDeviceSynchronize();
        hipEventElapsedTime(&elapsed_time_ms1, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(i >= 100)time1 += elapsed_time_ms1;
    }
    printf("normal:%f\n", time1);

    signed char res[32*32*16];
    hipMemcpy(res, d_char_out, sizeof( signed char) * 32*32*16, hipMemcpyDeviceToHost);
    

    //Measure
    float time2 = 0;
    for(int i=0; i<1000; i++){
        padding<<<16, dim3(32,32)>>>(d_char, d_charp);
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        hipMemset(&d_char_outp, 0, sizeof(signed char)*(34*34*16));
		winograd<<<dim3(16, 16), dim3(4,4,16)>>>(d_charp, d_wino, d_char_outp);
        elapsed_time_ms2=0.0f;
        hipEventRecord(stop, 0);
        hipDeviceSynchronize();
        hipEventElapsedTime(&elapsed_time_ms2, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(i >= 100)time2 += elapsed_time_ms2;
    } 
    printf("winograd:%f\n", time2);
    
    signed char res1[34*34*16];
    hipMemcpy(res1, d_char_outp, sizeof(signed char) * 34*34*16, hipMemcpyDeviceToHost);

    //check data
    signed char resp[PSIZE] = {0};
    for(int i=0;i<16;i++){
        for (int j=0;j<32; j++){
            for (int k=0;k<32; k++){
                resp[j+1 + (k+1)*34 + i*1156] = res[j + k*32 + i*1024];
            }
        }
    }

    int miss = 0;
    for(int i=0;i<PSIZE; i++) if(resp[i] != res1[i]) {printf("%d ", i); miss++;}
    // for(int i=0;i<PSIZE; i++) if(resp[i] != res1[i]) {miss++;}
    if(miss == 0) printf("%f 倍速くなりました。", time1/time2);
    else if(miss != 0) printf("bat!");

    return;
    
}