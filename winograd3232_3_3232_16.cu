#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include <iostream>
#include <random>

#define SIZE  (32*32*3)
#define OUTSIZE  (32*32*16)
#define PSIZE  (34*34*3)
#define POUTSIZE  (34*34*16)
#define FSIZE  (3*3*3*16)
#define WSIZE  (4*4*3*16)

void initialData( signed char *a, int size){
    for (int i = 0; i < size; i++) a[i] =  signed char(i);
    return;
}

__device__  signed char clamp(int v)
{
	if(v <= -128) return -128;
	else if(v > 127) return 127;
    else return v;
}

__global__ void conv( signed char *input,  signed char *filter,  signed char *output){
    // input = (32,16,3), output =(32,16,16), filter =(3,3,3,16), s_input = (34,18,3)

    const int id = threadIdx.x + blockDim.x*threadIdx.y, ch_id = blockIdx.x, block_size = blockDim.x*blockDim.y;
    const int conv_place = (ch_id&1), conv_ch = (ch_id>>1);
    
    __shared__ signed char s_input[1836]; //34*18*3
	__shared__ signed char s_filter[27]; //3*3*3
    __shared__ int s_output[512];  //32*16
    
    // init shared memory
    for(int i = id; i < 1836; i+=block_size) s_input[i] = 0;
    for(int j = id; j < 27; j+=block_size) s_filter[j] = filter[j + conv_ch*27];
    for(int k = id; k < 512; k+=block_size) s_output[k] = 0;

    for (int i = id; i < 1632; i+=blockDim.x){   //1632 = 32*(16+1)*3,   544 = 32*(16+1)
        const int x = i&31, y = ((i%544)>>5), ch = i/(544);
        s_input[(x+1) + 34*(y+(conv_place^1)) + 612*ch] = input[x + ((y + (conv_place<<4) - conv_place)<<5) + (ch<<10)];
    }__syncthreads();

    for (int n = id; n < 1536; n += block_size){
        const int x = n&31;
        const int y = (n&511)>>5;
        const int ch = n>>9;
        const int x0 = s_input[(x)  +34*(y)  + 612*ch] * s_filter[0 + 9*ch];
        const int x1 = s_input[(x+1)+34*(y)  + 612*ch] * s_filter[1 + 9*ch];
        const int x2 = s_input[(x+2)+34*(y)  + 612*ch] * s_filter[2 + 9*ch];
        const int x3 = s_input[(x)  +34*(y+1)+ 612*ch] * s_filter[3 + 9*ch];
        const int x4 = s_input[(x+1)+34*(y+1)+ 612*ch] * s_filter[4 + 9*ch];
        const int x5 = s_input[(x+2)+34*(y+1)+ 612*ch] * s_filter[5 + 9*ch];
        const int x6 = s_input[(x)  +34*(y+2)+ 612*ch] * s_filter[6 + 9*ch];
        const int x7 = s_input[(x+1)+34*(y+2)+ 612*ch] * s_filter[7 + 9*ch];
        const int x8 = s_input[(x+2)+34*(y+2)+ 612*ch] * s_filter[8 + 9*ch];
        atomicAdd(&s_output[x+(y<<5)], x0+x1+x2+x3+x4+x5+x6+x7+x8);
    }
    __syncthreads();
    for (int i = id; i < 512; i+=blockDim.x){
        const int x = i&31, y = (i>>5) + (conv_place<<4);
        output[x + (y<<5) + (conv_ch<<10)] = ((s_output[i] + (1 << 4)) >> 5) + 128;
    } 
}


__global__ void winograd( signed char *input,  signed short *weight,  signed char  *output){
    const int id = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    const int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z, bx = blockIdx.x, by = blockIdx.y;
	const int in_start = bx*2 + tx + (by*2+ty)*34 + tz*1156;  //1156 = 34*34
	
    

	__shared__ signed char input_smem [3][4][4];
	__shared__ int Btd [3][4][4];
	__shared__ int BtdB [3][4][4];
	__shared__ int I [16][4][4];
    __shared__ int AtI [16][2][4];
	
	I[tz][ty][tx] = 0;
	input_smem[tz][ty][tx] = input[in_start];
	// __syncthreads();
	switch (ty)
	{
	case 0:
		Btd [tz][ty][tx] = input_smem[tz][tx][0] - input_smem[tz][tx][2];
		break;
	case 1:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 2:
		Btd [tz][ty][tx] = - input_smem[tz][tx][1] + input_smem[tz][tx][2];
		break;
	case 3:
		Btd [tz][ty][tx] = input_smem[tz][tx][1] - input_smem[tz][tx][3];
		break;
	}
	// __syncthreads();
	switch (tx)
	{
	case 0:
		BtdB[tz][tx][ty] = Btd[tz][ty][0] - Btd[tz][ty][2];
		break;
	case 1:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 2:
		BtdB[tz][tx][ty] = - Btd[tz][ty][1] + Btd[tz][ty][2];
		break;
	case 3:
		BtdB[tz][tx][ty] = Btd[tz][ty][1] - Btd[tz][ty][3];
		break;
	}
	// __syncthreads();
	for(int i=id; i<48*16; i+=48){
        const int ch = i/48;
		atomicAdd(&I[ch][ty][tx], BtdB[tz][ty][tx]*weight[i]);
	}

    __syncthreads();
    const int temp = tx + (ty<<2);
    if(tz == 0 && temp <16) {
        const int out_start1 = (bx*2+1) + ((by*2+1)*34) + ((temp)*1156);
        const int out_start2 = (bx*2+2) + ((by*2+1)*34) + ((temp)*1156);
        const int out_start3 = (bx*2+1) + ((by*2+2)*34) + ((temp)*1156);
        const int out_start4 = (bx*2+2) + ((by*2+2)*34) + ((temp)*1156);
        output[out_start1] = clamp((((I[temp][0][0] + I[temp][0][1] + I[temp][0][2] + I[temp][1][0] + I[temp][1][1] + I[temp][1][2] + I[temp][2][0] + I[temp][2][1] + I[temp][2][2]) + (1 << 6)) >>7)) + 128;
        output[out_start2] = clamp((((I[temp][0][1] - I[temp][0][2] - I[temp][0][3] + I[temp][1][1] - I[temp][1][2] - I[temp][1][3] + I[temp][2][1] - I[temp][2][2] - I[temp][2][3]) + (1 << 6)) >>7)) + 128;
        output[out_start3] = clamp((((I[temp][1][0] + I[temp][1][1] + I[temp][1][2] - I[temp][2][0] - I[temp][2][1] - I[temp][2][2] - I[temp][3][0] - I[temp][3][1] - I[temp][3][2]) - (1 << 6)) >>7)) + 128;
        output[out_start4] = clamp((((I[temp][1][1] - I[temp][1][2] - I[temp][1][3] - I[temp][2][1] + I[temp][2][2] + I[temp][2][3] - I[temp][3][1] + I[temp][3][2] + I[temp][3][3]) + (1 << 6)) >>7)) + 128;
    }

}


__global__ void padding( signed char *input,  signed char *output){
    const int id  = threadIdx.x + blockDim.x*threadIdx.y;
    const int idx = threadIdx.x;
    const int idy = threadIdx.y;
    const int ch = blockIdx.x;
    __shared__  signed char s_output[34*34];

    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) s_output[i] = 0;
    __syncthreads();
    s_output[(idx+1) + (idy+1)*34] = input[idx + (idy<<5) + (ch<<10)];
    __syncthreads();
    for(int i=id; i< 34*34; i+=blockDim.x*blockDim.y) output[i + ch*34*34] = s_output[i];
}


int main(){
    hipEvent_t start, stop;
    float elapsed_time_ms;
     signed char *h_char = ( signed char *)malloc(SIZE * sizeof( signed char));

    initialData(h_char, SIZE);

    // allocate global memory
    signed char *d_char, *d_char_out, *d_char_outp, *d_charp, *d_filter;
    signed short *d_wino;
    hipMalloc( (void **) &d_char, SIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_out, OUTSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_char_outp, POUTSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_charp, PSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_filter, FSIZE * sizeof( signed char) );
    hipMalloc( (void **) &d_wino, WSIZE * sizeof( signed short) );

    hipMemcpy( d_char, h_char, SIZE * sizeof( signed char), hipMemcpyHostToDevice );

    signed char f;
    signed short f_short;
	FILE* fp;
    signed char x1_1[FSIZE];
    signed short wino[WSIZE];
    fp = fopen( "./conv_block.conv.weight", "rb" );
    if (!fp) printf("x1_1: pathを間違えています\n");
    for(int i=0; i<FSIZE; i++){
        if( fread( &f, sizeof(f), 1, fp ) < 1 ){
            fputs( "x1_1: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        x1_1[i] = f;
    }
    hipMemcpy(d_filter, x1_1, sizeof( signed char) * FSIZE, hipMemcpyHostToDevice);
    if (fp) fclose(fp);


    fp = fopen( "./wino_params_short/conv_block.conv.weight", "rb" );
    if (!fp) printf("wino: pathを間違えています\n");
    for(int i=0; i<WSIZE; i++){
        if( fread( &f_short, sizeof(f_short), 1, fp ) < 1 ){
            fputs( "wino: 読み込み中にエラーが発生しました。\n", stderr );
            exit( EXIT_FAILURE );
        }
        wino[i] = f_short;
    }
    hipMemcpy(d_wino, wino, sizeof(signed short) * WSIZE, hipMemcpyHostToDevice);
    if (fp) fclose(fp);
    
    
    //Measure load store uint8
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int i=0; i<1000; i++) {
        // hipMemset(&d_char_outp, 0, sizeof(signed char)*PSIZE);
        conv<<<32, 256>>>(d_char, d_filter, d_char_out);
    }
    elapsed_time_ms=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("normal:%f\n", elapsed_time_ms);

    signed char res[OUTSIZE];
    hipMemcpy(res, d_char_out, sizeof(signed char) * OUTSIZE, hipMemcpyDeviceToHost);
    for(int i=0; i<32; i++) printf("%d, ", res[i]);
    printf("\n");

    //Measure load store uint8
    padding<<<3, dim3(32,32)>>>(d_char, d_charp);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int i=0; i<1000; i++){
        hipMemset(&d_char_outp, 0, sizeof(signed char)*PSIZE);
		winograd<<<dim3(16, 16), dim3(4,4,3)>>>(d_charp, d_wino, d_char_outp);
    } 
    elapsed_time_ms=0.0f;
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("winograd:%f\n", elapsed_time_ms);
    
    signed char res1[POUTSIZE];
    hipMemcpy(res1, d_char_outp, sizeof(signed char) * POUTSIZE, hipMemcpyDeviceToHost);
    for(int i=35; i<68; i++) printf("%d, ", char (res1[i]));
    printf("\n");


    free(h_char );
    hipFree(d_char);
    hipFree(d_char_out);
    hipFree(d_char_outp);
    hipFree(d_charp);
    hipFree(d_filter);
    hipFree(d_wino);

    return;
    
}